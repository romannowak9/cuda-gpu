#include "hip/hip_runtime.h"
#include "mult_mv.h"

__global__ void multMatrixVector(float *b, float *A, float *x, unsigned int nrows, unsigned int ncols)
{
    const int row = threadIdx.x + blockIdx.x * blockDim.x;
    // Inside full context space
    if (row < nrows) {
        float sum = 0.0f;
        const unsigned int base = row * ncols;
        for (unsigned int col = 0; col < ncols; ++col)
            sum += A[base + col] * x[col];
            
        b[row] = sum;
    }
}

Matrix multMatrixVectorOnDevice(const Matrix &A, const Matrix &x)
{
    if (A.getCols() != x.getRows())
    {
        throw std::runtime_error("Matrix and vector dimensions do not match for multiplication.");
    }

    Matrix outMatrix(A.getRows(), x.getCols());

    // allocate input and output in the device
    float *d_A;
    float *d_x;
    float *d_outMatrix;

    hipMalloc((void **)&d_A, A.getRows() * A.getCols() * sizeof(float));
    hipMalloc((void **)&d_x, x.getRows() * x.getCols() * sizeof(float));
    hipMalloc((void **)&d_outMatrix, outMatrix.getRows() * outMatrix.getCols() * sizeof(float));

    // copy to the device
    hipMemcpy(d_A, A.getDataConstPtr(), A.getRows() * A.getCols() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x.getDataConstPtr(), x.getRows() * x.getCols() * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 1024;  // Maksimum block size on device
    int gridSize = (A.getRows() + blockSize - 1) / blockSize;

    multMatrixVector<<<gridSize, blockSize>>>(d_outMatrix, d_A, d_x, A.getRows(), A.getCols());

    hipMemcpy(outMatrix.getDataPtr(), d_outMatrix, outMatrix.getRows() * outMatrix.getCols() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_outMatrix);

    return outMatrix;
}

Matrix multMatrixVectorOnHost(const Matrix &A, const Matrix &x)
{
    if (A.getCols() != x.getRows())
    {
        throw std::runtime_error("Matrix and vector dimensions do not match for multiplication.");
    }

    Matrix b(A.getRows(), 1);
    for (unsigned int i = 0; i < A.getRows(); ++i)
    {
        float sum = 0.0f;
        for (unsigned int j = 0; j < A.getCols(); ++j)
        {
            sum += A.getDataConstPtr()[i * A.getCols() + j] * x.getDataConstPtr()[j];
        }
        b.getDataPtr()[i] = sum;
    }
    return b;
}
